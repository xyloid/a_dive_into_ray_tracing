// #include "obj_parser.h"


#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <sstream>
#include <string>
#include <vector>

int main() {
  // vector<triangle> triangles;

  std::string filename = "objs/dafault_cube_in_triangles.obj";

  // std::ifstream infile("objs/test.obj");
  std::ifstream infile(filename);

  if (infile.is_open()) {
    std::string line;
    while (std::getline(infile, line)) {
      std::cout << line << std::endl;
    }
    infile.close();
  } else {
  
    std::cerr << "read failed" << std::endl;
  }
  return 0;
}